#include "hip/hip_runtime.h"
#include "../error.cuh"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

using namespace cooperative_groups;

const int REPEATS = 10;
const int N = 150000000;
const int M = sizeof(real) * N;
const int BLOCK_SIZE = 128;
const int GRID_SIZE_1 = 10240;
//must be less than (N + BLOCK_SIZE - 1) / BLOCK_SIZE to make an opt.

void timing(real *h_x, real *d_x);

int main()
{
    real *h_x = (real *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));

    printf("Using static shared memory, the cooperative group with optimization:\n");
    timing(h_x, d_x);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ reduce_opt(real *d_x, real *d_y,const int N){
 
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ real s_y[BLOCK_SIZE]; 
    //Logically this should be blockDim.x, which is grammarly forbiddened.
    //Use dynamic memory instead to solve this.

    real y = 0.0;//no conflict in the first loop.
    const int stride = gridDim.x * blockDim.x;
    for(int n = bid * blockDim.x + tid; n < N; n += stride){
        y += d_x[n];
    }
    s_y[tid] = y;
    __syncthreads();

    for(int offset = blockDim.x >> 1; offset >= 32; offset >>= 1){
        if(tid < offset){
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads(); //make sure that a round have been completed
    }
    
    y = s_y[tid];
    thread_block_tile<32>g = tiled_partition<32>(this_thread_block());
    for(int ofs = 16; ofs > 0; ofs >>= 1){  
        // To continue reducing in [0,32), instead of blockDim.x >> 1, which turns out with
        // a result 3 times larger.
        y += g.shfl_down(y, ofs);
    }

    //after all rounds are completed, pick it if it is the first item
    if(tid == 0){
        d_y[bid] = y;
    }
}

real reduce(real *d_x){

    real *d_y;
    const int ymem = sizeof(real) * GRID_SIZE_1;
    CHECK(hipMalloc(&d_y, ymem));
    reduce_opt<<<GRID_SIZE_1, BLOCK_SIZE>>>(d_x, d_y, N);

    real *d_z = new real;
    CHECK(hipMalloc(&d_z, sizeof(real)));
    reduce_opt<<<1, BLOCK_SIZE>>>(d_y, d_z, GRID_SIZE_1);

    real *h_z = new real;
    CHECK(hipMemcpy(h_z, d_z, sizeof(real), hipMemcpyDeviceToHost));
    real result = *h_z;

    CHECK(hipFree(d_y));
    free(h_z);
    CHECK(hipFree(d_z));
    return result;
}

void timing(real *h_x, real *d_x){

    real sum = 0;

    float t_sum = 0;
    float t2_sum = 0;
  
    for(int rep = 0; rep <= REPEATS; rep ++){

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
        sum = reduce(d_x);  
        //Inv-cpy is included, which differs according to the method.

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed;
        CHECK(hipEventElapsedTime(&elapsed, start, stop));
    
        if(rep > 0){
            t_sum += elapsed;
            t2_sum += elapsed * elapsed;
        }
    }

    const float t_ave = t_sum / REPEATS;
    const float t_err = sqrt(t2_sum/REPEATS - t_ave*t_ave);
    printf("Time = %g +- %g ms\n",t_ave, t_err);  //and \n !
    printf("sum = %f.\n", sum);
    printf("\n");
}
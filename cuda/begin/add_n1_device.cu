
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

const double EPS = 1.0e-15;
const double a = 1.2;
const double b = 2.3;
const double c = 3.5;

void __global__ add(const double *x, const double *y, double *z, const int n);
void check(const double *z, const int N);

int main(){
    const int n = 10000000 + 1;  //10^7 + 1
    const int M = sizeof(double) * n;
    double *h_x = new double[n];
    double *h_y = new double[n];
    double *h_z = new double[n];

    for(int i=0;i<n;i++){
        h_x[i] = a;
        h_y[i] = b;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);
    
    const int block_size = 128;
    const int grid_size = (n-1)/block_size + 1;   //make sure it is enough
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, n);

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);//cudaMemcpyDeviceToHost
    check(h_z, n);

    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    hipFree(d_x);//cudaFree
    hipFree(d_y);
    hipFree(d_z);

    hipDeviceSynchronize();
    return 0;
}

double __device__ add_device(const double x, const double y){
    return x + y;
}

void __global__ add(const double *x, const double *y, double *z, const int n){
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n){
        z[i] = add_device(x[i], y[i]);
    }
    if(blockIdx.x == 78125)printf("threadIdx.x: %d, z[i] = %.5f\n",threadIdx.x, z[i]);
}

void check(const double *z, const int n){
    bool has_err = false;
    for(int i=0; i < n; i++){
        if(fabs(z[i]-c) > EPS){
            has_err = true;
        }
    }
    printf("%s\n", has_err ? "Has_errors" : "No error");
}
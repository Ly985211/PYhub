#include "hip/hip_runtime.h"
#include "error.cuh"
#include<stdio.h>
#include<math.h>

const double EPS = 1.0e-15;
const double a = 1.2;
const double b = 2.3;
const double c = 3.5;

void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main(){
    const int n = 10000000;  //10^7
    const int M = sizeof(double) * n;
    double *h_x = new double[n];
    double *h_y = new double[n];
    double *h_z = new double[n];

    for(int i=0;i<n;i++){
        h_x[i] = a;
        h_y[i] = b;
    }

    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice)); //hipMemcpyHostToDevice
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice)); //hipMemcpyHostToDevice
    
    const int block_size = 1280;
    const int grid_size = n / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);
    
    CHECK(hipGetLastError());//check the kernel
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));//hipMemcpyDeviceToHost
    check(h_z, n); // if put after"delete[]",no output

    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    CHECK(hipFree(d_x));//hipFree
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));

    
    return 0;
}

void __global__ add(const double *x, const double *y, double *z){
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    z[i] = x[i] + y[i];
    
    if(i%500000==0)printf("blockIdx.x=%d,threadIdx.x=%d, x = %.5f,y = %.5f, z = %.5f\n",blockIdx.x, threadIdx.x, x[i], y[i], z[i]);
}

void check(const double *z, const int n){
    bool has_err = false;
    for(int i=0; i < n; i++){
        if(fabs(z[i]-c) > EPS){
            has_err = true;
        }
    }
    printf("%s\n", has_err ? "Has_errors" : "No error");
}